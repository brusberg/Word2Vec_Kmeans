#include "hip/hip_runtime.h"
#ifndef LAUNCHES_H_
#define LAUNCHES_H_
#include <float.h>
#define THREADS_PER_BLOCK 512

int * cpuGOLD(int n, int k, int iter, int dim, double * data, double * centroids){
  // Label 1, Label 2, Label 3, .....
  int *labels = (int*)malloc(n*sizeof(int));
  if (!labels) {
    fprintf(stderr, "labels unable to allocate %d ints\n", n);
    return NULL; // fail but return
  }

  // Size of centroid 1, size of centroid 2
  int *sizes = (int*)malloc(k*sizeof(int));
  if (!sizes) {
    fprintf(stderr, "sizes unable to allocate %d ints\n", dim);
    return NULL; // fail but return
  }
 
  //Distance N=0/Centroid = 0, Distance N=0/Centroid = 1, ..., Distance N=0/Centroid = k, Distance N=1/Centroid = 0, .....
  double* distances = (double*)malloc(n*k*sizeof(double));
  if (!distances) {
    fprintf(stderr, "distances  unable to allocate %d x %d doubles\n", n, k);
    return NULL; // fail but return
  }

  /*TWO STEP PROCESS
   *Classify
   *Find Means */
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    for(int vector = 0; vector < n; vector++){
      for(int centroid = 0; centroid < k; centroid++){
	double distance = 0;
	for(int dimension = 0; dimension < dim; dimension++){
	  double temp = centroids[centroid*dim+dimension] - data[vector*dim+dimension];
	  distance += temp*temp;
	}
	distance = sqrt(distance);
	distances[(vector*k)+centroid] = distance;
      }
    }
    
    //Reset sizes
    for(int i = 0;i < k; i ++){
      sizes[i] = 1;
    }
    
    //Classify each vector
    for(int vector = 0; vector < n; vector++){
      int minIndex = 0;
      double min = DBL_MAX;
      
      for(int centroid = 0; centroid < k; centroid++){
	if( distances[(vector*k)+centroid] < min){
	  minIndex = centroid;
	  min = distances[(vector*k)+centroid];
	}
      }
      labels[vector] = minIndex;
    }
    
    //Go through each vector and update the mean of the centroid it is classified as
    for(int vector = 0; vector < n; vector++){
      sizes[labels[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroids[labels[vector]*dim+d] = (centroids[labels[vector]*dim+d] * (sizes[labels[vector]]-1) + data[vector*dim+d]) / sizes[labels[vector]];
      }
    }
    //Iterate
    b++;
  }
  //{v1,v1,v2,v2,v3,v3}
  // 0  1  2  3  4  5
  for(int vector = 0; vector < n; vector++){
    int minIndex = 0;
    double min = DBL_MAX;
    
    for(int centroid = 0; centroid < k; centroid++){
      if( distances[(vector*k)+centroid] < min){
	minIndex = centroid;
	min = distances[(vector*k)+centroid];
      }
    }
    labels[vector] = minIndex;
    //printf("%d,%d,%d\n",(int)data[vector*dim],(int)data[vector*dim+1], minIndex);
    //printf("%d,%d\n",vector, minIndex); 
  }
  free(sizes);
  free(distances);
  return labels;
}

int * launchVectorDistance(int * labelsGPU, double * data, double *distances,double * centroidsGPU, int *sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));
  
  double *distancesD, *centroidsD, *dataD;
  hipError_t code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    hipMemcpy(centroidsD, centroidsGPU, k * dim * sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(distancesD, distances, n * k * sizeof(double), hipMemcpyHostToDevice);
    
    vectorDistance<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n);

    hipMemcpy(distances, distancesD, n * k * sizeof(double), hipMemcpyDeviceToHost);

 
    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    //Classify each vecotr with its closeset centroid
    for(int vector = 0; vector < n; vector++){
      int minIndex = 0;
      double min = DBL_MAX;
      
      for(int centroid = 0; centroid < k; centroid++){
	if( distances[(vector*k)+centroid] < min){
	  minIndex = centroid;
	  min = distances[(vector*k)+centroid];
	}
      }
      labelsGPU[vector] = minIndex;
    }

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //iterate
    b++;
  }//END WHILE

   
  
  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  
  return labelsGPU;
}

int * launchVectorDistanceUR(int * labelsGPU, double * data, double *distances,double * centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));
  
  double *distancesD, *centroidsD, *dataD;
  hipError_t code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    hipMemcpy(centroidsD, centroidsGPU, k * dim * sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(distancesD, distances, n * k * sizeof(double), hipMemcpyHostToDevice);

    if(dim%8==0){
      vectorDistance2UR8<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n);
    }else if(dim%4==0){
      vectorDistance2UR4<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n);
    }else if(dim%2==0){
      vectorDistance2UR2<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n);
    }else{
      vectorDistance2<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n);
    }
    hipMemcpy(distances, distancesD, n * k * sizeof(double), hipMemcpyDeviceToHost);

 
    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    //Classify each vecotr with its closeset centroid
    for(int vector = 0; vector < n; vector++){
      int minIndex = 0;
      double min = DBL_MAX;
      
      for(int centroid = 0; centroid < k; centroid++){
	if( distances[(vector*k)+centroid] < min){
	  minIndex = centroid;
	  min = distances[(vector*k)+centroid];
	}
      }
      labelsGPU[vector] = minIndex;
    }

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //iterate
    b++;
  }//END WHILE

   
  
  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  
  return labelsGPU;
}

int * launchCentroidDistanceUR(int * labelsGPU, double * data, double *distances, double* centroidsGPU, int *sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  double *distancesD, *centroidsD, *dataD;
  
  hipError_t code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    hipMemcpy(centroidsD, centroidsGPU, k * dim * sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(distancesD, distances, n * k * sizeof(double), hipMemcpyHostToDevice);
    //DO i actually need to copy over distances?
    for(int centroid = 0; centroid < k; centroid ++){
      if(dim%8==0){
	centroidDistance2UR8<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, centroid, n);
      }else if(dim%4==0){	
	centroidDistance2UR4<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, centroid, n);
      }else if(dim%2==0){
	centroidDistance2UR2<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, centroid, n);
      }else{
	centroidDistance2<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, centroid, n);
      }
    }
    hipMemcpy(distances, distancesD, n * k * sizeof(double), hipMemcpyDeviceToHost);

    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    //Classify each vecotr with its closeset centroid
    for(int vector = 0; vector < n; vector++){
      int minIndex = 0;
      double min = DBL_MAX;
      
      for(int centroid = 0; centroid < k; centroid++){
	if( distances[(vector*k)+centroid] < min){
	  minIndex = centroid;
	  min = distances[(vector*k)+centroid];
	}
      }
      labelsGPU[vector] = minIndex;
    }

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //iterate
    b++;
  }//END WHILE

   

  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  return labelsGPU;
}

int * launchCentroidDistance(int * labelsGPU, double * data, double *distances, double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  double *distancesD, *centroidsD, *dataD;
  
  hipError_t code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    hipMemcpy(centroidsD, centroidsGPU, k * dim * sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(distancesD, distances, n * k * sizeof(double), hipMemcpyHostToDevice);
    //DO i actually need to copy over distances?
    for(int centroid = 0; centroid < k; centroid ++){
      centroidDistance<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, centroid, n);
    }
    hipMemcpy(distances, distancesD, n * k * sizeof(double), hipMemcpyDeviceToHost);

    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    //Classify each vecotr with its closeset centroid
    for(int vector = 0; vector < n; vector++){
      int minIndex = 0;
      double min = DBL_MAX;
      
      for(int centroid = 0; centroid < k; centroid++){
	if( distances[(vector*k)+centroid] < min){
	  minIndex = centroid;
	  min = distances[(vector*k)+centroid];
	}
      }
      labelsGPU[vector] = minIndex;
    }

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //iterate
    b++;
  }//END WHILE

   

  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  return labelsGPU;
}

int * launchCentroidConstantDistanceUR(int * labelsGPU, double * data, double *distances, double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  double *distancesD, *dataD;
  
  hipError_t code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    for(int centroid = 0; centroid < k; centroid ++){
      hipMemcpyToSymbol(HIP_SYMBOL(centroidCONST),  centroidsGPU+centroid*dim, sizeof(double)*dim);
      if(dim%8==0){
	centroidConstantDistance2UR8<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
      }else if(dim%4==0){	
	centroidConstantDistance2UR4<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
      }else if(dim%2==0){
	centroidConstantDistance2UR2<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
      }else{
	centroidConstantDistance2<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
      }

    }
    hipMemcpy(distances, distancesD, n * k * sizeof(double), hipMemcpyDeviceToHost);
    
    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    //Classify each vecotr with its closeset centroid
    for(int vector = 0; vector < n; vector++){
      int minIndex = 0;
      double min = DBL_MAX;
      
      for(int centroid = 0; centroid < k; centroid++){
	if( distances[(vector*k)+centroid] < min){
	  minIndex = centroid;
	  min = distances[(vector*k)+centroid];
	}
      }
      labelsGPU[vector] = minIndex;
    }

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //iterate
    b++;
  }//END WHILE

   
  hipFree(distancesD);
  hipFree(dataD);
  return labelsGPU;
}

int * launchCentroidConstantDistance(int * labelsGPU, double * data, double *distances, double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  double *distancesD, *dataD;
  
  hipError_t code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    for(int centroid = 0; centroid < k; centroid ++){
      hipMemcpyToSymbol(HIP_SYMBOL(centroidCONST),  centroidsGPU+centroid*dim, sizeof(double)*dim);

      centroidConstantDistance<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
    }
    hipMemcpy(distances, distancesD, n * k * sizeof(double), hipMemcpyDeviceToHost);
    
    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    //Classify each vecotr with its closeset centroid
    for(int vector = 0; vector < n; vector++){
      int minIndex = 0;
      double min = DBL_MAX;
      
      for(int centroid = 0; centroid < k; centroid++){
	if( distances[(vector*k)+centroid] < min){
	  minIndex = centroid;
	  min = distances[(vector*k)+centroid];
	}
      }
      labelsGPU[vector] = minIndex;
    }

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //iterate
    b++;
  }//END WHILE

   
  hipFree(distancesD);
  hipFree(dataD);
  return labelsGPU;
}

int * launchCentroidConstantDistanceNSQR(int * labelsGPU, double * data, double *distances, double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  double *distancesD, *dataD;
  
  hipError_t code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    for(int centroid = 0; centroid < k; centroid ++){
      hipMemcpyToSymbol(HIP_SYMBOL(centroidCONST),  centroidsGPU+centroid*dim, sizeof(double)*dim);

      centroidConstantDistanceNSQR<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
    }
    hipMemcpy(distances, distancesD, n * k * sizeof(double), hipMemcpyDeviceToHost);
    
    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    //Classify each vecotr with its closeset centroid
    for(int vector = 0; vector < n; vector++){
      int minIndex = 0;
      double min = DBL_MAX;
      
      for(int centroid = 0; centroid < k; centroid++){
	if( distances[(vector*k)+centroid] < min){
	  minIndex = centroid;
	  min = distances[(vector*k)+centroid];
	}
      }
      labelsGPU[vector] = minIndex;
    }

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //iterate
    b++;
  }//END WHILE

   
  hipFree(distancesD);
  hipFree(dataD);
  return labelsGPU;
}

int * launchCentroidConstantDistanceNSQ(int * labelsGPU, double * data, double *distances, double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  double *distancesD, *dataD;
  
  hipError_t code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    //hipMemcpy(distancesD, distances, n * k * sizeof(double), hipMemcpyHostToDevice);
    //DO I actually need to copy over distances?
    for(int centroid = 0; centroid < k; centroid ++){
      hipMemcpyToSymbol(HIP_SYMBOL(centroidCONST),  centroidsGPU+centroid*dim, sizeof(double)*dim);

      centroidConstantDistanceNSQ<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
    }
    hipMemcpy(distances, distancesD, n * k * sizeof(double), hipMemcpyDeviceToHost);
    
    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    //Classify each vecotr with its closeset centroid
    for(int vector = 0; vector < n; vector++){
      int minIndex = 0;
      double min = DBL_MAX;
      
      for(int centroid = 0; centroid < k; centroid++){
	if( distances[(vector*k)+centroid] < min){
	  minIndex = centroid;
	  min = distances[(vector*k)+centroid];
	}
      }
      labelsGPU[vector] = minIndex;
    }

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //iterate
    b++;
  }//END WHILE

   
  hipFree(distancesD);
  hipFree(dataD);
  return labelsGPU;
}

int * launchLabelMins(int * labelsGPU, double * data, double *distances, double* centroidsGPU, int *sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));
  
  int* labelsD;
  double *distancesD;
  hipError_t code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"LabelsD GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"DistancesD GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(distancesD, distances, n * k * sizeof(double), hipMemcpyHostToDevice);
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    
    for(int vector = 0; vector < n; vector++){
      for(int centroid = 0; centroid < k; centroid++){
	double distance = 0;
	for(int dimension = 0; dimension < dim; dimension++){
	  double temp = centroidsGPU[centroid*dim+dimension] - data[vector*dim+dimension];
	  distance += temp*temp;
	}
	distance = sqrt(distance);
	distances[(vector*k)+centroid] = distance;
      }
    }
 
    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }
    
    hipMemcpy(distancesD, distances, n * k * sizeof(double), hipMemcpyHostToDevice);
    labelMins1<<<dimGrid, dimBlock>>>(labelsD, distancesD, k, n);
    hipMemcpy(labelsGPU, labelsD, n * sizeof(int), hipMemcpyDeviceToHost);
    //Classify each vecotr with its closeset centroid
    for(int vector = 0; vector < n; vector++){
      int minIndex = 0;
      double min = DBL_MAX;
      
      for(int centroid = 0; centroid < k; centroid++){
	if( distances[(vector*k)+centroid] < min){
	  minIndex = centroid;
	  min = distances[(vector*k)+centroid];
	}
      }
      labelsGPU[vector] = minIndex;
    }
    
    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //iterate
    b++;
  }//END WHILE
  
  hipFree(distancesD);
  hipFree(labelsD);
  
  return labelsGPU;
}

int * launchThreadPerVectorWithLabels(int * labelsGPU, double * data, double *distances,double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  int* labelsD;
  double *distancesD, *centroidsD, *dataD;
  hipError_t   code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    hipMemcpy(centroidsD, centroidsGPU, k * dim * sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(centroidsD, centroidsGPU, k * dim * sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(distancesD, distances, n * k * sizeof(double), hipMemcpyHostToDevice);
    
    vectorLabelDistance<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD);

    hipMemcpy(labelsGPU, labelsD, n * sizeof(int), hipMemcpyDeviceToHost);
    //hipMemcpy(distances, distancesD, n * k * sizeof(double), hipMemcpyDeviceToHost);

    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //iterate
    b++;
  }//END WHILE

   

  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  hipFree(labelsD);

  return labelsGPU;
}

int * launchThreadPerVectorWithLabelsUR(int * labelsGPU, double * data, double *distances,double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  int* labelsD;
  double *distancesD, *centroidsD, *dataD;
  hipError_t   code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    hipMemcpy(centroidsD, centroidsGPU, k * dim * sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(centroidsD, centroidsGPU, k * dim * sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(distancesD, distances, n * k * sizeof(double), hipMemcpyHostToDevice);

    if(dim%8==0){
      vectorLabelDistance1UR8<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD);
    }else if(dim%4==0){
      vectorLabelDistance1UR4<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD);
    }else if(dim%2==0){
      vectorLabelDistance1UR2<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD);
    }else{
      vectorLabelDistance1<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD);
    }
    hipMemcpy(labelsGPU, labelsD, n * sizeof(int), hipMemcpyDeviceToHost);
    //hipMemcpy(distances, distancesD, n * k * sizeof(double), hipMemcpyDeviceToHost);

    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //iterate
    b++;
  }//END WHILE

   

  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  hipFree(labelsD);

  return labelsGPU;
}

int * launchCentroidConstantLabels(int * labelsGPU, double * data, double *distances,double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  int* labelsD;
  double *distancesD, *centroidsD, *dataD;
  
  hipError_t   code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    for(int centroid = 0; centroid < k; centroid ++){
      hipMemcpyToSymbol(HIP_SYMBOL(centroidCONST),  (centroidsGPU+centroid*dim), sizeof(double)*dim);
      centroidConstantDistance<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
    }
    
    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    labelMins1<<<dimGrid, dimBlock>>>(labelsD, distancesD, k, n);
    hipMemcpy(labelsGPU, labelsD, n * sizeof(int), hipMemcpyDeviceToHost);

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //for(int d = 0; d < dim; d ++){
    //  fprintf(stderr, "centroidsGPU[%d]=%f\n", d, centroidsGPU[d]);
    // }
    //iterate
    b++;
  }//END WHILE

   
 

  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  hipFree(labelsD);

  return labelsGPU;
}

int * launchCentroidConstantLabelsUR(int * labelsGPU, double * data, double *distances,double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  int* labelsD;
  double *distancesD, *centroidsD, *dataD;
  
  hipError_t   code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    for(int centroid = 0; centroid < k; centroid ++){
      hipMemcpyToSymbol(HIP_SYMBOL(centroidCONST),  (centroidsGPU+centroid*dim), sizeof(double)*dim);
      if(dim%8==0){
	centroidConstantDistance2UR8<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
      }else if(dim%4==0){	
	centroidConstantDistance2UR4<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
      }else if(dim%2==0){
	centroidConstantDistance2UR2<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
      }else{
	centroidConstantDistance2<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
      }
    }
    hipMemcpy(distances, distancesD, n * k * sizeof(double), hipMemcpyDeviceToHost);

    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 1;
    }

    hipMemcpy(distancesD, distances, n * k * sizeof(double), hipMemcpyHostToDevice);
    labelMins1<<<dimGrid, dimBlock>>>(labelsD, distancesD, k, n);
    hipMemcpy(labelsGPU, labelsD, n * sizeof(int), hipMemcpyDeviceToHost);

    //Go through each vector and update the mean of the classified centroid
    for(int vector = 0; vector < n; vector++){
      sizes[labelsGPU[vector]] += 1;
      for(int d = 0; d < dim; d++){
	centroidsGPU[labelsGPU[vector]*dim+d] = (centroidsGPU[labelsGPU[vector]*dim+d] * (sizes[labelsGPU[vector]]-1) + data[vector*dim+d]) / sizes[labelsGPU[vector]];
      }
    }
    //for(int d = 0; d < dim; d ++){
    //  fprintf(stderr, "centroidsGPU[%d]=%f\n", d, centroidsGPU[d]);
    // }
    //iterate
    b++;
  }//END WHILE

   
 

  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  hipFree(labelsD);

  return labelsGPU;
}

int * launchUpdateMeans(int * labelsGPU, double * data, double *distances,double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  int *labelsD;
  double *centroidsD, *dataD;
  
  hipError_t   code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    for(int vector = 0; vector < n; vector++){
      for(int centroid = 0; centroid < k; centroid++){
	double distance = 0;
	for(int dimension = 0; dimension < dim; dimension++){
	  double temp = centroidsGPU[centroid*dim+dimension] - data[vector*dim+dimension];
	  distance += temp*temp;
	}
	distance = sqrt(distance);
	distances[(vector*k)+centroid] = distance;
      }
    }
    //Reset sizes
    for(int i = 0; i < k; i ++){
      sizes[i] = 0;
    }

    //Classify each vector with its closeset centroid
    for(int vector = 0; vector < n; vector++){
      int minIndex = 0;
      double min = DBL_MAX;
      
      for(int centroid = 0; centroid < k; centroid++){
	if( distances[(vector*k)+centroid] < min){
	  minIndex = centroid;
	  min = distances[(vector*k)+centroid];
	}
      }
      labelsGPU[vector] = minIndex;
      sizes[minIndex] += 1;
    }

    //Load Labels
    hipMemcpy(labelsD, labelsGPU, n * sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy(centroidsD, centroidsGPU, k * dim * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(centroidsD, 0, k * dim * sizeof(double));
    updateMeans1<<<dimGrid, dimBlock>>>(dataD, centroidsD, labelsD, dim, n);
    hipMemcpy(centroidsGPU, centroidsD, k * dim * sizeof(double), hipMemcpyDeviceToHost);
    //Load back sums
    
    for(int i = 0; i < k; i ++){
      double size = (double)sizes[i];
      for(int d = 0; d < dim; d ++){
	centroidsGPU[i*dim+d] = centroidsGPU[i*dim+d]/size;
      }
    }
    //for(int d = 0; d < dim; d ++){
    //  fprintf(stderr, "centroidsGPU[%d]=%f\n", d, centroidsGPU[d]);
    //}
    //iterate
    b++;
  }//END WHILE

   

  hipFree(centroidsD);
  hipFree(dataD);
  hipFree(labelsD);

  return labelsGPU;
}

int * launchGPUVector(int * labelsGPU, double * data, double *distances,double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  int *labelsD;
  int *sizesD;
  double  *distancesD, *centroidsD, *dataD;
  
  hipError_t  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&sizesD, k * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    hipMemcpy(centroidsD, centroidsGPU, k * dim * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(sizesD, 0, k * sizeof(int));
    if(dim%8==0){
      vectorLabelDistance1UR8<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
    }else if(dim%4==0){
      vectorLabelDistance1UR4<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
    }else if(dim%2==0){
      vectorLabelDistance1UR2<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
    }else{
      vectorLabelDistance1<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
    }
    
    hipMemset(centroidsD, 0, k * dim * sizeof(double));
    updateMeans1<<<dimGrid, dimBlock>>>(dataD, centroidsD, labelsD, dim, n);
    hipMemcpy(centroidsGPU, centroidsD, k * dim * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sizes, sizesD, k * sizeof(int), hipMemcpyDeviceToHost);
    //Load back sums
    
    for(int i = 0; i < k; i ++){
      double size = (double)sizes[i];
      for(int d = 0; d < dim; d ++){
	centroidsGPU[i*dim+d] = centroidsGPU[i*dim+d]/size;
      }
    }
    b++;
  }//END WHILE
  
  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  hipFree(labelsD);
  hipFree(sizesD);
  
  return labelsGPU;
}

int * launchGPUCentroid(int * labelsGPU, double * data, double *distances,double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));

  int *labelsD;
  int *sizesD;
  double  *distancesD, *centroidsD, *dataD;
  
  hipError_t  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&sizesD, k * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    
    for(int centroid = 0; centroid < k; centroid ++){
      hipMemcpyToSymbol(HIP_SYMBOL(centroidCONST),  (centroidsGPU+centroid*dim), sizeof(double)*dim);
      centroidConstantDistance<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
    }

    hipMemset(sizesD, 0, k * sizeof(int));
    labelMins1<<<dimGrid, dimBlock>>>(labelsD, distancesD, k, n, sizesD);  hipMemset(centroidsD, 0, k * dim * sizeof(double));
    updateMeans1<<<dimGrid, dimBlock>>>(dataD, centroidsD, labelsD, dim, n);
    hipMemcpy(centroidsGPU, centroidsD, k * dim * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sizes, sizesD, k * sizeof(int), hipMemcpyDeviceToHost);
    //Load back sums
    
    for(int i = 0; i < k; i ++){
      double size = (double)sizes[i];
      for(int d = 0; d < dim; d ++){
	centroidsGPU[i*dim+d] = centroidsGPU[i*dim+d]/size;
      }
    }
    b++;
  }//END WHILE
  
  return labelsGPU;
}

/* Used as a helper for the > 512 case
   Given the number of elements it returns the minimum x in the expression 512^x > number of elements */ 
int fastSteps(int size)
{
  int steps = 0;
  unsigned int mySize = 1;
  while (mySize < size)
    {
      mySize <<= 9;
      steps++;
    }
  return steps;
}

int * launchReduction(int * labelsGPU, double * data, double *distances,double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));
  
  int *labelsD;
  int *sizesD;
  double  *distancesD, *centroidsD, *dataD;
  double *bufferD;
  hipError_t  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&bufferD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&sizesD, k * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  int blockSize = 512;
  //Calculate the number of blocks needed
  int num_blocks = n / blockSize;
  if (n % blockSize != 0){
    num_blocks++;
  }
  //Calculates the number of iterations needed of kernel calls
  int steps;
  //Gives the spacing between adjacent elements at each step of the iteration
  
  int level;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    hipMemset(sizesD, 0, k * sizeof(int));
    if(dim%8==0){
      vectorLabelDistance2UR8<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
    }else if(dim%4==0){
      vectorLabelDistance2UR4<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
    }else if(dim%2==0){
      vectorLabelDistance2UR2<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
    }else{
      vectorLabelDistance2<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
    }

    
    for(int i = 0; i < k; i++){
      reduceToBuffer<<<dimGrid, dimBlock>>>(dataD, bufferD, labelsD, dim, n, k);
      level = 1;
      steps = fastSteps(n);
      //fprintf(stderr,"Fuck %d %d\n", b, i);
      while(steps--){
	//fprintf(stderr,"Fuck %d\n", steps);
	//This takes the array, number of elements, and what each index has to be multiplied by
	reduce<<<num_blocks, blockSize>>>(bufferD, centroidsD, labelsD, dim, n, k, level, sizesD);
	//Multiply the level by 512
	level <<= 9;
	//Calculate the number blocks needed for the next iteration
	int temp = num_blocks;
	num_blocks /= blockSize;
	if (temp % blockSize != 0){
	  num_blocks++;
	}
      }
    }
    //At this points we have coorect labels and sizes
    //We want to reduce into buffer
    //We then want to reduce buffer
    //   and divide by sizes
    //So we can update buffers
    b++;
  }//END WHILE
  hipMemcpy(labelsGPU, labelsD, n * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(centroidsGPU, centroidsD, n * sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(bufferD);
  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  hipFree(labelsD);
  hipFree(sizesD);
  
  return labelsGPU;
}

//add const

int * launchReductionConst(int * labelsGPU, double * data, double *distances,double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));
  
  int *labelsD;
  int *sizesD;
  double  *distancesD, *centroidsD, *dataD;
  double *bufferD;
  hipError_t  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&bufferD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&sizesD, k * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  int blockSize = 512;
  //Calculate the number of blocks needed
  int num_blocks = n / blockSize;
  if (n % blockSize != 0){
    num_blocks++;
  }
  //Calculates the number of iterations needed of kernel calls
  int steps;
  //Gives the spacing between adjacent elements at each step of the iteration
  
  int level;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    for(int centroid = 0; centroid < k; centroid ++){
      hipMemcpyToSymbol(HIP_SYMBOL(centroidCONST),  (centroidsGPU+centroid*dim), sizeof(double)*dim);
      centroidConstantDistance<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
    }
    
    labelMins1<<<dimGrid,dimBlock>>>(labelsD, distancesD, k, n, sizesD);

    for(int i = 0; i < k; i++){
      hipMemset(sizesD, 0, k * sizeof(int));
      reduceToBuffer<<<dimGrid, dimBlock>>>(dataD, bufferD, labelsD, dim, n, k);
      level = 1;
      steps = fastSteps(n);
      //fprintf(stderr,"Fuck %d %d\n", b, i);
      while(steps--)
        {
	  //fprintf(stderr,"Fuck %d\n", steps);
	  //This takes the array, number of elements, and what each index has to be multiplied by
	  reduce<<<num_blocks, blockSize>>>(bufferD, centroidsD, labelsD, dim, n, k, level, sizesD);
	  //Multiply the level by 512
	  level <<= 9;
	  //Calculate the number blocks needed for the next iteration
	  int temp = num_blocks;
	  num_blocks /= blockSize;
	  if (temp % blockSize != 0){
	    num_blocks++;
	  }
        }
      
    }
    //iterate
    b++;
  }//END WHILE
  hipMemcpy(labelsGPU, labelsD, n * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(centroidsGPU, centroidsD, n * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(bufferD);
  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  hipFree(labelsD);
  hipFree(sizesD);
  
   
  return labelsGPU;
}

int * launchReductionUR(int * labelsGPU, double * data, double *distances,double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));
  
  int *labelsD;
  int *sizesD;
  double  *distancesD, *centroidsD, *dataD;
  double *bufferD;
  hipError_t  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&bufferD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&sizesD, k * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  int blockSize = 512;
  //Calculate the number of blocks needed
  int num_blocks = n / blockSize;
  if (n % blockSize != 0){
    num_blocks++;
  }
  //Calculates the number of iterations needed of kernel calls
  int steps;
  //Gives the spacing between adjacent elements at each step of the iteration
  
  int level;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    hipMemset(sizesD, 0, k * sizeof(int));
    if(dim%8==0){
      vectorLabelDistance2UR8<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
      for(int i = 0; i < k; i++){
	reduceToBufferUR8<<<dimGrid, dimBlock>>>(dataD, bufferD, labelsD, dim, n, k);
	level = 1;
	steps = fastSteps(n);
	//fprintf(stderr,"Fuck %d %d\n", b, i);
	while(steps--){
	  //fprintf(stderr,"Fuck %d\n", steps);
	  //This takes the array, number of elements, and what each index has to be multiplied by
	  reduceUR8<<<num_blocks, blockSize>>>(bufferD, centroidsD, labelsD, dim, n, k, level, sizesD);
	  //Multiply the level by 512
	  level <<= 9;
	  //Calculate the number blocks needed for the next iteration
	  int temp = num_blocks;
	  num_blocks /= blockSize;
	  if (temp % blockSize != 0){
	    num_blocks++;
	  }
	}
      }
    }else if(dim%4==0){
      vectorLabelDistance2UR4<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
      for(int i = 0; i < k; i++){
	reduceToBufferUR4<<<dimGrid, dimBlock>>>(dataD, bufferD, labelsD, dim, n, k);
	level = 1;
	steps = fastSteps(n);
	//fprintf(stderr,"Fuck %d %d\n", b, i);
	while(steps--){
	  //fprintf(stderr,"Fuck %d\n", steps);
	  //This takes the array, number of elements, and what each index has to be multiplied by
	  reduceUR4<<<num_blocks, blockSize>>>(bufferD, centroidsD, labelsD, dim, n, k, level, sizesD);
	  //Multiply the level by 512
	  level <<= 9;
	  //Calculate the number blocks needed for the next iteration
	  int temp = num_blocks;
	  num_blocks /= blockSize;
	  if (temp % blockSize != 0){
	    num_blocks++;
	  }
	}
      }
    }else if(dim%2==0){
      vectorLabelDistance2UR2<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
      for(int i = 0; i < k; i++){
	reduceToBufferUR2<<<dimGrid, dimBlock>>>(dataD, bufferD, labelsD, dim, n, k);
	level = 1;
	steps = fastSteps(n);
	//fprintf(stderr,"Fuck %d %d\n", b, i);
	while(steps--){
	  //fprintf(stderr,"Fuck %d\n", steps);
	  //This takes the array, number of elements, and what each index has to be multiplied by
	  reduceUR2<<<num_blocks, blockSize>>>(bufferD, centroidsD, labelsD, dim, n, k, level, sizesD);
	  //Multiply the level by 512
	  level <<= 9;
	  //Calculate the number blocks needed for the next iteration
	  int temp = num_blocks;
	  num_blocks /= blockSize;
	  if (temp % blockSize != 0){
	    num_blocks++;
	  }
	}
      }
    }else{
      vectorLabelDistance2<<<dimGrid, dimBlock>>>(dataD, centroidsD, distancesD, k, dim, n, labelsD, sizesD);
      for(int i = 0; i < k; i++){
	reduceToBuffer<<<dimGrid, dimBlock>>>(dataD, bufferD, labelsD, dim, n, k);
	level = 1;
	steps = fastSteps(n);
	//fprintf(stderr,"Fuck %d %d\n", b, i);
	while(steps--){
	  //fprintf(stderr,"Fuck %d\n", steps);
	  //This takes the array, number of elements, and what each index has to be multiplied by
	  reduce<<<num_blocks, blockSize>>>(bufferD, centroidsD, labelsD, dim, n, k, level, sizesD);
	  //Multiply the level by 512
	  level <<= 9;
	  //Calculate the number blocks needed for the next iteration
	  int temp = num_blocks;
	  num_blocks /= blockSize;
	  if (temp % blockSize != 0){
	    num_blocks++;
	  }
	}
      }
    }    
    //At this points we have coorect labels and sizes
    //We want to reduce into buffer
    //We then want to reduce buffer
    //   and divide by sizes
    //So we can update buffers
    b++;
  }//END WHILE
  hipMemcpy(labelsGPU, labelsD, n * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(centroidsGPU, centroidsD, n * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(bufferD);
  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  hipFree(labelsD);
  hipFree(sizesD);
  
  return labelsGPU;
}

//add const

int * launchReductionConstUR(int * labelsGPU, double * data, double *distances,double* centroidsGPU, int * sizes, int n, int k, int iter, int dim){
  dim3 dimBlock(THREADS_PER_BLOCK);
  dim3 dimGrid(ceil((float)n/(float)THREADS_PER_BLOCK));
  
  int *labelsD;
  int *sizesD;
  double  *distancesD, *centroidsD, *dataD;
  double *bufferD;
  hipError_t  code = hipMalloc(&centroidsD, k * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&distancesD, n * k * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&dataD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&bufferD, n * dim * sizeof(double));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&labelsD, n * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  code = hipMalloc(&sizesD, k * sizeof(int));
  if(code!=hipSuccess){
    fprintf(stderr,"GPUassert: %s\n", hipGetErrorName(code));
  }
  hipMemcpy(dataD,data, n * dim * sizeof(double), hipMemcpyHostToDevice);
  
  int b = 0;
  int blockSize = 512;
  //Calculate the number of blocks needed
  int num_blocks = n / blockSize;
  if (n % blockSize != 0){
    num_blocks++;
  }
  //Calculates the number of iterations needed of kernel calls
  int steps;
  //Gives the spacing between adjacent elements at each step of the iteration
  
  int level;
  while(b < iter){
    //for each vector
    //find the distance to each centroid
    for(int centroid = 0; centroid < k; centroid ++){
      hipMemcpyToSymbol(HIP_SYMBOL(centroidCONST),  (centroidsGPU+centroid*dim), sizeof(double)*dim);
      centroidConstantDistance<<<dimGrid, dimBlock>>>(dataD, distancesD, k, dim, centroid, n);
    }
    
    labelMins1<<<dimGrid,dimBlock>>>(labelsD, distancesD, k, n, sizesD);

    if(dim%8==0){
      for(int i = 0; i < k; i++){
	hipMemset(sizesD, 0, k * sizeof(int));
	reduceToBufferUR8<<<dimGrid, dimBlock>>>(dataD, bufferD, labelsD, dim, n, k);
	level = 1;
	steps = fastSteps(n);
	//fprintf(stderr,"Fuck %d %d\n", b, i);
	while(steps--)
	  {
	    //fprintf(stderr,"Fuck %d\n", steps);
	    //This takes the array, number of elements, and what each index has to be multiplied by
	    reduceUR8<<<num_blocks, blockSize>>>(bufferD, centroidsD, labelsD, dim, n, k, level, sizesD);
	    //Multiply the level by 512
	    level <<= 9;
	    //Calculate the number blocks needed for the next iteration
	    int temp = num_blocks;
	    num_blocks /= blockSize;
	    if (temp % blockSize != 0){
	      num_blocks++;
	    }
	  }
      
      }
    }else if(dim%4==0){
      for(int i = 0; i < k; i++){
	hipMemset(sizesD, 0, k * sizeof(int));
	reduceToBufferUR4<<<dimGrid, dimBlock>>>(dataD, bufferD, labelsD, dim, n, k);
	level = 1;
	steps = fastSteps(n);
	//fprintf(stderr,"Fuck %d %d\n", b, i);
	while(steps--)
	  {
	    //fprintf(stderr,"Fuck %d\n", steps);
	    //This takes the array, number of elements, and what each index has to be multiplied by
	    reduceUR4<<<num_blocks, blockSize>>>(bufferD, centroidsD, labelsD, dim, n, k, level, sizesD);
	    //Multiply the level by 512
	    level <<= 9;
	    //Calculate the number blocks needed for the next iteration
	    int temp = num_blocks;
	    num_blocks /= blockSize;
	    if (temp % blockSize != 0){
	      num_blocks++;
	    }
	  }
      
      }
    }else if(dim%2==0){
      for(int i = 0; i < k; i++){
	hipMemset(sizesD, 0, k * sizeof(int));
	reduceToBufferUR2<<<dimGrid, dimBlock>>>(dataD, bufferD, labelsD, dim, n, k);
	level = 1;
	steps = fastSteps(n);
	//fprintf(stderr,"Fuck %d %d\n", b, i);
	while(steps--)
	  {
	    //fprintf(stderr,"Fuck %d\n", steps);
	    //This takes the array, number of elements, and what each index has to be multiplied by
	    reduceUR2<<<num_blocks, blockSize>>>(bufferD, centroidsD, labelsD, dim, n, k, level, sizesD);
	    //Multiply the level by 512
	    level <<= 9;
	    //Calculate the number blocks needed for the next iteration
	    int temp = num_blocks;
	    num_blocks /= blockSize;
	    if (temp % blockSize != 0){
	      num_blocks++;
	    }
	  }
      
      }
    }else{
      for(int i = 0; i < k; i++){
	hipMemset(sizesD, 0, k * sizeof(int));
	reduceToBuffer<<<dimGrid, dimBlock>>>(dataD, bufferD, labelsD, dim, n, k);
	level = 1;
	steps = fastSteps(n);
	//fprintf(stderr,"Fuck %d %d\n", b, i);
	while(steps--)
	  {
	    //fprintf(stderr,"Fuck %d\n", steps);
	    //This takes the array, number of elements, and what each index has to be multiplied by
	    reduce<<<num_blocks, blockSize>>>(bufferD, centroidsD, labelsD, dim, n, k, level, sizesD);
	    //Multiply the level by 512
	    level <<= 9;
	    //Calculate the number blocks needed for the next iteration
	    int temp = num_blocks;
	    num_blocks /= blockSize;
	    if (temp % blockSize != 0){
	      num_blocks++;
	    }
	  }
      
      }
    }
    
    //iterate
    b++;
  }//END WHILE
  hipMemcpy(labelsGPU, labelsD, n * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(centroidsGPU, centroidsD, n * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(bufferD);
  hipFree(distancesD);
  hipFree(centroidsD);
  hipFree(dataD);
  hipFree(labelsD);
  hipFree(sizesD);
  
   
  return labelsGPU;
}


#endif // #ifndef LAUNCHES_H_
